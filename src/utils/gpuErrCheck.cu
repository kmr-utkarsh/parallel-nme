#include "gpuErrCheck.h"

inline void gpuAssert(hipError_t code, int line) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUAssert: %s %s %dn", hipGetErrorString(code), line);
      		exit(code);
	}
}
